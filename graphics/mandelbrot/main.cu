
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <time.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
      file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

__global__ void color_pixel(int* pixels, int x_size, int y_size, int max_iterations){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= x_size) || (j >= y_size)) return;  // make sure we're actually operating on the array

  float c_re = (i - x_size/2.0f)*4.0f/x_size;
  float c_im = (j - y_size/2.0f)*4.0f/x_size;
  float x = 0, y = 0;
  int iteration = 0;
  while (x*x+y*y <= 4 && iteration < max_iterations) {
    float x_new = x*x - y*y + c_re;
    y = 2*x*y + c_im;
    x = x_new;
    iteration++;
  }

  int pixel_index = j * x_size + i;
  if (iteration < max_iterations) {
    pixels[pixel_index] = iteration;
  }
  else {
    pixels[pixel_index] = 0;
  }
}

int main(int argc, char *argv[]){
  int nx = 1000;
  int max = 1000;
  int tx = 8;  // number of thread blocks 
  int ty = 8;  // number of threads in a block

  if(argc > 1){
    nx = atoi(argv[1]);
  }

  int ny = nx;   // for now, grid must be square, so ny = nx

  //allocate color values array
  int num_pixels = nx * ny;
  size_t pixels_size = num_pixels * sizeof(int);

  int *pixels;
  checkCudaErrors(hipMallocManaged((void **)&pixels, pixels_size));  // allocate shared memory 

  clock_t start, stop;
  start = clock();

  dim3 blocks(nx/tx+1,ny/ty+1); // 1221 blocks 
  dim3 threads(tx,ty);  // total of tx * ty (64) threads in a block
  color_pixel<<<blocks, threads>>>(pixels, nx, ny, max);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  std::cerr << "took " << timer_seconds << " seconds.\n";

  std::cout << "P3\n" << nx << " " << ny << "\n255\n";
  for(int x = 0; x < nx; x++){
    for(int y = 0; y < ny; y++){
      int pixel_index = x * nx + y;

      int r = 0; // (img[x][y] * 255/100);
      int g = (pixels[pixel_index] * 255/10);
      int b = 0; //(img[x][y] * 255);
      std::cout << r << " " <<  g << " " << b << "\n";
    }
  }
}
