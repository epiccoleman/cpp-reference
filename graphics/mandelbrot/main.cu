
#include <hip/hip_runtime.h>
#include <iostream>
#include <iostream>
#include <cmath>
#include <vector>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
      file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

__global__ void color_pixel(int* pixels, int x_size, int y_size, int max_iterations){

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= x_size) || (j >= y_size)) return;

  double c_re = (i - x_size/2.0)*4.0/x_size;
  double c_im = (j - y_size/2.0)*4.0/x_size;
  double x = 0, y = 0;
  int iteration = 0;
  while (x*x+y*y <= 4 && iteration < max_iterations) {
    double x_new = x*x - y*y + c_re;
    y = 2*x*y + c_im;
    x = x_new;
    iteration++;
  }

  int pixel_index = j * x_size + i;
  if (iteration < max_iterations) {
    pixels[pixel_index] = iteration;
  }
  else {
    pixels[pixel_index] = 0;
  }
}

int main(int argc, char *argv[]){
  int nx = 1200;
  int max = 1000;
  int tx = 8;
  int ty = 8;

  if(argc > 1){
    nx = atoi(argv[1]);
  }

  int ny = nx;

  //allocate color values array
  int num_pixels = nx * ny;
  size_t pixels_size = num_pixels * sizeof(int);

  int *pixels;
  checkCudaErrors(hipMallocManaged((void **)&pixels, pixels_size));

  dim3 blocks(nx/tx+1,ny/ty+1);
  dim3 threads(tx,ty);
  color_pixel<<<blocks, threads>>>(pixels, nx, ny, max);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  std::cout << "P3\n" << nx << " " << ny << "\n255\n";
  for(int x = 0; x < nx; x++){
    for(int y = 0; y < ny; y++){
      int pixel_index = x * nx + y;

      int r = 0; // (img[x][y] * 255/100);
      int g = (pixels[pixel_index] * 255/10);
      int b = 0; //(img[x][y] * 255);
      std::cout << r << " " <<  g << " " << b << "\n";
    }
  }
}
